#include "hip/hip_runtime.h"
﻿#include "device_types.h"

#include "CudaEcsSystem.h"
#define __HIPCC__
#include <mma.h>
using namespace nvcuda;
// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void wmma_example(half* a, half* b, float* c,
                             int M, int N, int K,
                             float alpha, float beta)
{
	// Leading dimensions. Packed with no transpositions.
	int lda = M;
	int ldb = K;
	int ldc = M;

	// Tile using a 2D grid
	int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
	// Declare the fragments
	wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
	wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
	wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
	wmma::fill_fragment(acc_frag, 0.0f);
	// Loop over the K-dimension
	for (int i = 0; i < K; i += WMMA_K)
	{
		int aRow = warpM * WMMA_M;
		int aCol = i;
		int bRow = i;
		int bCol = warpN * WMMA_N;

		// Bounds checking
		if (aRow < M && aCol < K && bRow < K && bCol < N)
		{
			// Load the inputs
			wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
			wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

			// Perform the matrix multiplication
			wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
		}
	} // Load in current value of c, scale by beta, and add to result scaled by alpha
	int cRow = warpM * WMMA_M;
	int cCol = warpN * WMMA_N;

	if (cRow < M && cCol < N)
	{
		wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

		for (int i = 0; i < c_frag.num_elements; i++)
		{
			c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
		} // Store the output
		wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
	}
}
