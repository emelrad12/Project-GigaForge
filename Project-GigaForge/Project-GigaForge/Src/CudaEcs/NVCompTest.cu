#include "hip/hip_runtime.h"
﻿#include "nvcomp/cascaded.hpp"
#include "nvcomp/lz4.hpp"
#include "nvcomp.hpp"
#include <cassert>
#include <cstdlib>
#include <vector>
#include "CudaGlobals.h"
#include "../../Timer.h"
#include "../CudaCommon/Bitset.h"
#define CUDA_CHECK checkCudaError
#define REQUIRE(x) if(!(x)){throw "Error";}else{}
using namespace std;
using namespace nvcomp;
#define Lz4
#undef Lz4
void PrintMb(std::string message, float size)
{
	std::cout << message << ": " << size / 1024.0f / 1024 << " MB" << std::endl;
}

template <typename T>
void test_lz4(T* input, int uncompressedSize, const size_t chunk_size = 1 << 16)
{
	hipFree(0);//Takes 250ms to initialize
	auto uncompressedBytes = uncompressedSize * sizeof(T);
	PrintMb("Data uncompressedSize", uncompressedBytes);
	Timer t = Timer();
	// create GPU only input buffer
	T* d_in_data;
	const size_t in_bytes = sizeof(T) * uncompressedSize;
	CUDA_CHECK(hipMalloc((void**)&d_in_data, in_bytes));
	t.Restart("Allocate on gpu");
	CUDA_CHECK(hipMemcpy(d_in_data, input, in_bytes, hipMemcpyHostToDevice));
	t.Restart("Copy to device");
	hipStream_t stream;
	hipStreamCreate(&stream);

	size_t comp_temp_bytes = 0;
	size_t comp_out_bytes = 0;
	void* d_comp_temp;
	void* d_comp_out;

#ifdef Lz4
	LZ4Compressor compressor(chunk_size);
#else
	CascadedCompressor compressor(TypeOf<T>(), 1, 0, false);
#endif

	compressor.configure(in_bytes, &comp_temp_bytes, &comp_out_bytes);
	REQUIRE(comp_temp_bytes > 0);
	REQUIRE(comp_out_bytes > 0);

	// allocate temp buffer
	CUDA_CHECK(hipMalloc(&d_comp_temp, comp_temp_bytes));

	// Allocate output buffer
	CUDA_CHECK(hipMalloc(&d_comp_out, comp_out_bytes));

	size_t* comp_out_bytes_ptr;
	hipMalloc((void**)&comp_out_bytes_ptr, sizeof(size_t));
	CUDA_CHECK(hipDeviceSynchronize());
	t.Restart("Intermediate");
	compressor.compress_async(d_in_data, in_bytes, d_comp_temp, comp_temp_bytes, d_comp_out, comp_out_bytes_ptr, stream);
	CUDA_CHECK(hipStreamSynchronize(stream));
	auto compressTime = t.Restart("Compress");
	CUDA_CHECK(hipMemcpy(
		&comp_out_bytes,
		comp_out_bytes_ptr,
		sizeof(comp_out_bytes),
		hipMemcpyDeviceToHost));
	hipFree(comp_out_bytes_ptr);

	PrintMb("Out data uncompressedSize", comp_out_bytes);
	std::cout << "Ratio: " << (float)uncompressedBytes / comp_out_bytes << std::endl;
	PrintMb("CompressThoughput", (float)uncompressedBytes / (compressTime / 1000.0f));
	hipFree(d_comp_temp);
	hipFree(d_in_data);
	// Test to make sure copying the compressed file is ok
	void* copied = 0;
	CUDA_CHECK(hipMalloc(&copied, comp_out_bytes));
	CUDA_CHECK(hipMemcpy(copied, d_comp_out, comp_out_bytes, hipMemcpyDeviceToDevice));
	hipFree(d_comp_out);
	d_comp_out = copied;

#ifdef Lz4
	LZ4Decompressor decompressor;
#else
	CascadedDecompressor decompressor;
#endif

	size_t decomp_temp_bytes;
	size_t decomp_out_bytes;
	decompressor.configure(
		d_comp_out,
		comp_out_bytes,
		&decomp_temp_bytes,
		&decomp_out_bytes,
		stream);

	void* d_decomp_temp;
	hipMalloc(&d_decomp_temp, decomp_temp_bytes);

	T* out_ptr;
	hipMalloc(&out_ptr, decomp_out_bytes);

	// make sure the data won't match input if not written to, so we can verify
	// correctness
	hipMemset(out_ptr, 0, decomp_out_bytes);
	CUDA_CHECK(hipStreamSynchronize(stream))
	t.Restart("Before decompress");
	decompressor.decompress_async(
		d_comp_out,
		comp_out_bytes,
		d_decomp_temp,
		decomp_temp_bytes,
		out_ptr,
		decomp_out_bytes,
		stream);
	CUDA_CHECK(hipStreamSynchronize(stream))
	auto decompressTime = t.Restart("Decompress");
	PrintMb("DecompressThoughput", (float)uncompressedBytes / (decompressTime / 1000.0f));


	// Copy result back to host
	T* res = new T[uncompressedSize];
	hipMemcpy(res, out_ptr, uncompressedSize * sizeof(T), hipMemcpyDeviceToHost);
	t.Restart("Copy to host");

	// Verify correctness
	// REQUIRE(res == input);
	for (size_t i = 0; i < uncompressedSize; i++)
	{
		REQUIRE(res[i] == input[i]);
	}

	hipFree(d_comp_out);
	hipFree(out_ptr);
	hipFree(d_decomp_temp);
	delete res;
}

void TestComp()
{
	auto t = Timer();
	using T = bool;
	int size = 1 << 30;
	auto bitset = GigaEntity::Bitset(size);
#pragma omp parallel for
	for (int i = 0; i < size; i++)
	{
		auto bo = sin(i / 100) > 0;
		auto rand = std::rand();
		if (rand < RAND_MAX / 50) bo = true;
		bitset.unpackedData[i] = bo;
	}
	bitset.Pack();
	test_lz4(bitset.packedData, bitset.packedSize);
	// test_lz4(bitset.unpackedData, bitset.unpackedSize);
	t.Stop("total");
}
