#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "device_types.h"
#include <cstdio>

#include "CudaEcsSystem.h"
#include "../Ecs/CommandBuffer.h"
#include "../Ecs/EntityManager.h"
#define ompLoop omp parallel for schedule(static, 5000 * 100)

using namespace GigaEntity;
__global__ void cuda_hello()
{
	printf("Hello World from GPU!\n");
}


__device__ void LambdaFunc(int entityIndex, int& item, int arguments)
{
	if (item > entityIndex)
	{
		item--;
	}
	else
	{
		item++;
	}
}

CreateKernelWithFunction(LambdaFunc, Kernel, int, int)

void CudaTest()
{
	auto manager = EntityManager();
	manager.AddType<int>();
	auto buffer = CommandBuffer();
	buffer.RegisterComponent<int>();
	constexpr auto count = DEBUG ? 5000 * 100 : 5000 * 10000;

	auto& handle = buffer.GetFastAddHandle<int>();
	for (int offset = 0; offset < count; offset += chunkSize)
	{
#pragma ompLoop
		for (int i = 0; i < chunkSize; i++)
		{
			const auto index = i + offset;
			if (index < count)
			{
				buffer.AddComponent<int>(index, index, handle);
			}
		}
	}

	manager.ExecuteCommands(buffer);

	auto cudaManager = CudaEntityManager(manager);
	cudaManager.CopyToCuda<int>();
	
	RunKernel(cudaManager, 0);
	hipDeviceSynchronize();
	checkCudaLastError
}
